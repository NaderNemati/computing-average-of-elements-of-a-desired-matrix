#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <numeric>
#include<time.h>
#define col 20000
#define row 10000

using namespace std;

static const int N = col * row;
static const int blockSize = 1024;
static const int gridSize = 24;

__global__ void MatrixAvg(const int* gArr, int arraySize, int* gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize / 2; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] += shArr[thIdx + size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}


int main()
{
    srand(time(0));
    int **arr;
    int *dev_arr;

    arr = new int *[row];
    arr[0] = new int [N];
    for (int i = 1; i < col;i++)
    {
        arr[i] = arr[i - 1] + col;
    }
    hipMalloc((void**)&dev_arr, N * sizeof(int));
    hipMemcpy(dev_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    for (int i = 0; i < N; i++)
    {
        arr[0][i] = rand();
    }

    int out;
    int *dev_out;
    hipMalloc((void**)&dev_out, sizeof(int) * gridSize);

    MatrixAvg << <gridSize, blockSize >> > (dev_arr, N, dev_out);
    //dev_out now holds the partial result
    MatrixAvg << <1, blockSize >> > (dev_out, gridSize, dev_out);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();

    hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
    cout << "Average is:" << out/N << endl;
    hipFree(dev_arr);
    hipFree(dev_out);
    return 0;
}
