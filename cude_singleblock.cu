#include "hip/hip_runtime.h"

#include <iostream>
#include <numeric>
#include <time.h>
#define col 48
#define row 30

using namespace std;

__global__ void MatrixAvg(float* input)
{
	int tid = threadIdx.x;
	auto step_size = 1;
	int threads_num = blockDim.x;

	while (threads_num > 0)
	{
		if (tid < threads_num)
		{
			const auto fst = tid * step_size * 2;
			const auto snd = fst + step_size;
			input[fst] += input[snd];
		}

		step_size <<= 1;
		threads_num >>= 1;
	}
}

int main()
{
  srand(time(0));
	int N = col * row;
	float** arr, * dev_arr;
	arr = new float * [row];
	arr[0] = new float [N];
	for (int i = 1; i < col;i++)
	{
		arr[i] = arr[i - 1] + col;
	}

	hipMalloc(&dev_arr, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		arr[0][i] = rand();
	}

	hipMemcpy(dev_arr, arr[0], N * sizeof(float), hipMemcpyHostToDevice);

	int blocksize = 1024;
	int gridsize = 24;

	MatrixAvg <<<gridsize,blocksize>>>(dev_arr);
	int out;
	hipMemcpy(&out, dev_arr, N * sizeof(float), hipMemcpyDeviceToHost);

	cout << "Average is:" << out/N << endl;

	hipFree(dev_arr);
	delete(arr[0]);
	delete(arr);

	return 0;
}
